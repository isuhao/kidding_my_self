#include "hip/hip_runtime.h"
/*
 *	Copyright (C) 2007, 2008, 2009, 2010, 2011, 2012. PARP Research Group.
 *	<http://perception.inf.um.es>
 *	University of Murcia, Spain.
 *
 *	This file is part of the QVision library.
 *
 *	QVision is free software: you can redistribute it and/or modify
 *	it under the terms of the GNU Lesser General Public License as
 *	published by the Free Software Foundation, version 3 of the License.
 *
 *	QVision is distributed in the hope that it will be useful,
 *	but WITHOUT ANY WARRANTY; without even the implied warranty of
 *	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *	GNU Lesser General Public License for more details.
 *
 *	You should have received a copy of the GNU Lesser General Public
 *	License along with QVision. If not, see <http://www.gnu.org/licenses/>.
 */

/// @file qvcudaipD.cu
/// @brief File from the QVision library.
/// @author PARP Research Group. University of Murcia, Spain.

#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "qvcudaipD.h"
#include "qvcudautils.h"

#define WARP_SIZE	16

#define LOWPASS5_DX	160
#define LOWPASS5_DY	16

#define ROW_TILE_W	160
#define COLUMN_TILE_W	16
#define COLUMN_TILE_H	48

__device__ __constant__ float d_Kernel[CUDA_MAX_MASK_SIZE];

///////////////////////////////////////////////////////////////////////////////
// Kernel function for subtracting two images:
///////////////////////////////////////////////////////////////////////////////
template<typename T> __global__ void Subtract(T *d_Data1, T *d_Data2, T *d_Result, int width, int height,int step)
{
  const int x = __mul24(blockIdx.x, 16) + threadIdx.x;
  const int y = __mul24(blockIdx.y, 16) + threadIdx.y;
  int p = __mul24(y, step/sizeof(T)) + x;
  if (x<width && y<height)
    d_Result[p] = d_Data1[p] - d_Data2[p];
  __syncthreads();
}

void Subtract_kernel(float *src1, float *src2, float *dst, int w, int h, int step)
{
	dim3 blocks(iDivUp(w,16),iDivUp(h,16));
	dim3 threads(16,16);
	Subtract<float><<<blocks,threads>>>(src1, src2, dst, w, h, step);
}

/*void Subtract_kernel(int *src1, int *src2, int *dst, int w, int h)
{
	dim3 blocks(iDivUp(w,16),iDivUp(h,16));
	dim3 threads(16,16);
	Subtract<int><<<blocks,threads>>>(src1, src2, dst, w, h);
}*/

///////////////////////////////////////////////////////////////////////////////
// Kernel function to lowpass filter and subsample image
///////////////////////////////////////////////////////////////////////////////
__global__ void ScaleDown(const float *d_Data, float *d_Result, int width, int height, int step_s, int step_d)
{
	__shared__ float inrow[LOWPASS5_DX+4]; // 164 elements, for 160(bloq_siz_X) + 2 margin on each
						// side; here we read rows from original image.
	__shared__ float brow[5*(LOWPASS5_DX/2)]; // 400 elements, 5 rows of 80 columns each;
						// here we store intermediate results of row filtering.
	__shared__ int yRead[LOWPASS5_DY+4], yWrite[LOWPASS5_DY+4];
						// To store indexes for each thread to read from source
						// and write to destination.
	#define dx2 (LOWPASS5_DX/2)
	const int tx = threadIdx.x;
	const int tx0 = tx+0*dx2;
	const int tx1 = tx+1*dx2;
	const int tx2 = tx+2*dx2;
	const int tx3 = tx+3*dx2;
	const int tx4 = tx+4*dx2;
	const int xStart = __mul24(blockIdx.x, LOWPASS5_DX);
	const int yStart = __mul24(blockIdx.y, LOWPASS5_DY);
	const int xWrite = xStart/2 + tx;
	const float *k = d_Kernel;
	if (tx<LOWPASS5_DY+4)
		{ // Threads 0-19 initialize arrays yRead e yWrite, of 20 positions each
		int y = yStart + tx - 2; // corrected by pedroe; Borjkman: int y = yStart + tx - 1;
		y = (y<0 ? 0 : y);
		y = (y>=height ? height-1 : y);
		yRead[tx] = __mul24(y, step_s/sizeof(float));
		yWrite[tx] = __mul24((yStart + tx - 4)/2, step_d/sizeof(float));
		}
	__syncthreads();

	int xRead = xStart + tx - WARP_SIZE;
	xRead = (xRead<0 ? 0 : xRead);
	xRead = (xRead>=width ? width-1 : xRead);
	for (int dy=0;dy<LOWPASS5_DY+4;dy+=5)
		{
		// dy=0,5,10,15 (unrolled loop, with 5 different steps by iteration):

		if (tx>=WARP_SIZE-2)
			{
			// Threads 14-177 (total 164) write 164 elements of inrow[]
			inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+0] + xRead];
			// We read in inrow 164 consecutive elements from rows 0 (1º for step), 5 (2º step),
			// 10 (3º step), 15 (4º step).
			}
		__syncthreads();
		if (tx<dx2)
		// Threads 0-79 compute row filter on first 80 elements of brow[]:
      			brow[tx0] = __fmul_rz(k[0],(inrow[2*tx]+inrow[2*tx+4])) + __fmul_rz(k[1],(inrow[2*tx+1]+inrow[2*tx+3])) + __fmul_rz(k[2],inrow[2*tx+2]);
		__syncthreads();
		if (tx<dx2 && dy>=4 && !(dy&1) && (xWrite<width/2) && (yWrite[dy+0]<height*step_d/2/sizeof(float)) )
			{
			// Finally, threads 0-79 compute column filter on 400 elements of brow[]
			// (which correspond to 5 rows of 80 elements each). But this is a little bit tricky!
			// It uses a segmentation technique: in fact we get into here only from the second step of
			// the for, when a complete brow[] packet of 400 has been computed yet:
			d_Result[yWrite[dy+0] + xWrite] = __fmul_rz(k[2],brow[tx3]) + __fmul_rz(k[0],brow[tx1]+brow[tx0]) + __fmul_rz(k[1],brow[tx2]+brow[tx4]);
			}

		// From here on, it is always the same. The only important difference is always in the third
		// if: observe that the alternate !(dy&1) and (dy&1) conditions are used to perform the row
		// writings in the destination image in a "one step yes, one step no" way:
		if (dy<(LOWPASS5_DY+3))
			{
			if (tx>=WARP_SIZE-2)
				inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+1] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx1] = __fmul_rz(k[0],(inrow[2*tx]+inrow[2*tx+4])) + __fmul_rz(k[1],(inrow[2*tx+1]+inrow[2*tx+3])) + __fmul_rz(k[2],inrow[2*tx+2]);
			__syncthreads();
			if (tx<dx2 && dy>=3 && (dy&1) && (xWrite<width/2) && (yWrite[dy+1]<height*step_d/2/sizeof(float)))
				d_Result[yWrite[dy+1] + xWrite] = __fmul_rz(k[2],brow[tx4]) + __fmul_rz(k[0],brow[tx2]+brow[tx1]) + __fmul_rz(k[1],brow[tx3]+brow[tx0]);
			}

		if (dy<(LOWPASS5_DY+2))
			{
			if (tx>=WARP_SIZE-2)
				inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+2] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx2] = __fmul_rz(k[0],(inrow[2*tx]+inrow[2*tx+4])) + __fmul_rz(k[1],(inrow[2*tx+1]+inrow[2*tx+3])) + __fmul_rz(k[2],inrow[2*tx+2]);
			__syncthreads();
			if (tx<dx2 && dy>=2 && !(dy&1) && (xWrite<width/2) && (yWrite[dy+2]<height*step_d/2/sizeof(float)))
				d_Result[yWrite[dy+2] + xWrite] = __fmul_rz(k[2],brow[tx0]) + __fmul_rz(k[0],brow[tx3]+brow[tx2]) + __fmul_rz(k[1],brow[tx4]+brow[tx1]);
			}

		if (dy<(LOWPASS5_DY+1))
			{
			if (tx>=WARP_SIZE-2)
				inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+3] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx3] = __fmul_rz(k[0],(inrow[2*tx]+inrow[2*tx+4])) +  __fmul_rz(k[1],(inrow[2*tx+1]+inrow[2*tx+3])) + __fmul_rz(k[2],inrow[2*tx+2]);
			__syncthreads();
			if (tx<dx2 && dy>=1 && (dy&1) && (xWrite<width/2) && (yWrite[dy+3]<height*step_d/2/sizeof(float)))
				d_Result[yWrite[dy+3] + xWrite] = __fmul_rz(k[2],brow[tx1]) + __fmul_rz(k[0],brow[tx4]+brow[tx3]) + __fmul_rz(k[1],brow[tx0]+brow[tx2]);
			}

		if (dy<LOWPASS5_DY)
			{
			if (tx>=WARP_SIZE-2)
				inrow[tx-WARP_SIZE+2] = d_Data[yRead[dy+4] + xRead];
			__syncthreads();
			if (tx<dx2)
				brow[tx4] = __fmul_rz(k[0],(inrow[2*tx]+inrow[2*tx+4])) + __fmul_rz(k[1],(inrow[2*tx+1]+inrow[2*tx+3])) + __fmul_rz(k[2],inrow[2*tx+2]);
			__syncthreads();
			if (tx<dx2 && !(dy&1) && (xWrite<width/2) && (yWrite[dy+4]<height*step_d/2/sizeof(float)))
				d_Result[yWrite[dy+4] + xWrite] = __fmul_rz(k[2],brow[tx2]) + __fmul_rz(k[0],brow[tx0]+brow[tx4]) + __fmul_rz(k[1],brow[tx1]+brow[tx3]);
			}
		__syncthreads();

	} // for(dy)
}

void ScaleDown_kernel(const float *src, float *dest, float* h_Kernel, int w, int h, int step_s, int step_d)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, 5*sizeof(float)));
	dim3 blocks(iDivUp(w, LOWPASS5_DX), iDivUp(h, LOWPASS5_DY));
	dim3 threads(LOWPASS5_DX + WARP_SIZE + 2);
	ScaleDown<<<blocks, threads>>>(src,dest,w,h,step_s,step_d);
}

// Loop unrolling templates for convolution, needed for best performance (according to Bjorkman):
//template<int i> __device__ float ConvRow(float *data) { return data[i]*d_Kernel[i] + ConvRow<i-1>(data); }
//template<> __device__ float ConvRow<-1>(float *data) { return 0; }
//template<int i> __device__ float ConvCol(float *data) { return data[i*COLUMN_TILE_W]*d_Kernel[i] + ConvCol<i-1>(data); }
//template<> __device__ float ConvCol<-1>(float *data) { return 0; }

// Substituted by pedroe by simple __device_ function (slightly slower, but with parametrizable radius, 
// changeable in execution time):
__device__ float ConvRow(float *data, int cont)
{
	float acum = 0.0;
	for(int i=0;i<cont;i++)
		acum += data[i]*d_Kernel[i];
	return acum;
}

__device__ float ConvCol(float *data, int cont)
{
	float acum = 0.0;
	for(int i=0;i<cont;i++)
		acum += data[i*COLUMN_TILE_W]*d_Kernel[i];
	return acum;
}

///////////////////////////////////////////////////////////////////////////////
// Row convolution filter
///////////////////////////////////////////////////////////////////////////////
__global__ void ConvRowGPU(const float *d_Data, float *d_Result, int RADIUS, int width, int height)
{
	//Data cache
	//__shared__ float data[RADIUS+ROW_TILE_W+RADIUS+1];
	__shared__ float data[2*CUDA_MAX_MASK_SIZE+ROW_TILE_W+1];

	//Current tile and apron limits, relative to row start:
	const int tileStart = __mul24(blockIdx.x, ROW_TILE_W);

	//Row start index in d_Data[]:
	const int rowStart = __mul24(blockIdx.y, width);
	const int rowEnd = rowStart + width - 1;
	const int loadPos = threadIdx.x - WARP_SIZE + tileStart;
	const int smemPos = threadIdx.x - WARP_SIZE + RADIUS;

	//Set the entire data cache contents
	if (smemPos>=0) 
		{
		if (loadPos<0)
			data[smemPos] = d_Data[rowStart];
		else if (loadPos>=width) 
			data[smemPos] = d_Data[rowEnd];
		else
			data[smemPos] = d_Data[rowStart + loadPos];
		}

	__syncthreads();

	//Clamp tile and apron limits by image borders
	const int tileEnd = tileStart + ROW_TILE_W - 1;
	const int tileEndClamped = min(tileEnd, width - 1);
	const int writePos = tileStart + threadIdx.x;


	if (writePos <= tileEndClamped)
		{
		const int smemPos = threadIdx.x + RADIUS;
		d_Result[rowStart + writePos] = ConvRow(data + smemPos - RADIUS,2*RADIUS+1);
		}

	__syncthreads();
}

///////////////////////////////////////////////////////////////////////////////
// Column convolution filter
///////////////////////////////////////////////////////////////////////////////
__global__ void ConvColGPU(const float *d_Data, float *d_Result, int RADIUS, int width, int height, int smemStride, int gmemStride)
{
	// Data cache
	//__shared__ float data[COLUMN_TILE_W*(RADIUS + COLUMN_TILE_H + RADIUS+1)];
	__shared__ float data[COLUMN_TILE_W*(2*CUDA_MAX_MASK_SIZE+COLUMN_TILE_H+1)];

	// Current tile and apron limits, in rows
	const int tileStart = __mul24(blockIdx.y, COLUMN_TILE_H);
	const int tileEnd = tileStart + COLUMN_TILE_H - 1;
	const int apronStart = tileStart - RADIUS;
	const int apronEnd = tileEnd + RADIUS;
	
	// Current column index
	const int columnStart = __mul24(blockIdx.x, COLUMN_TILE_W) + threadIdx.x;
	const int columnEnd = columnStart + __mul24(height-1, width);

	if (columnStart<width)
		{
		// Shared and global memory indices for current column
		int smemPos = __mul24(threadIdx.y, COLUMN_TILE_W) + threadIdx.x;
		int gmemPos = __mul24(apronStart + threadIdx.y, width) + columnStart;
		// Cycle through the entire data cache
		for (int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y)
			{
			if (y<0)
				data[smemPos] = d_Data[columnStart];
			else if (y>=height)
				data[smemPos] = d_Data[columnEnd];
			else 
				data[smemPos] = d_Data[gmemPos];
			smemPos += smemStride;
			gmemPos += gmemStride;
			}
		}

	__syncthreads();

	if (columnStart<width)
		{
		// Shared and global memory indices for current column
		// Clamp tile and apron limits by image borders
		const int tileEndClamped = min(tileEnd, height - 1);
		int smemPos = __mul24(threadIdx.y + RADIUS, COLUMN_TILE_W) + threadIdx.x;
		int gmemPos = __mul24(tileStart + threadIdx.y , width) + columnStart;
		// Cycle through the tile body, clamped by image borders
		// Calculate and output the results
		for (int y=tileStart+threadIdx.y;y<=tileEndClamped;y+=blockDim.y)
			{
			d_Result[gmemPos] = ConvCol(data + smemPos - RADIUS*COLUMN_TILE_W,2*RADIUS+1);
			smemPos += smemStride;
			gmemPos += gmemStride;
			}
		}

	__syncthreads();
}

void SeparableFilter_kernel(const float *src, float *dest, float *temp, float *h_Kernel, int radius, int w, int h, int step)
{
	const unsigned int kernelSize = (2*radius+1);
	if(kernelSize > CUDA_MAX_MASK_SIZE) {
		std::cerr << "QVCUDAIP module: SeparableFilter_kernel function: kernelSize > CUDA_MAX_MASK_SIZE.\n";
		return;
	}
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, kernelSize*sizeof(float)));

	dim3 blockGridRows(iDivUp(w, ROW_TILE_W), h);
	dim3 threadBlockRows(WARP_SIZE + ROW_TILE_W + radius);

	ConvRowGPU<<<blockGridRows, threadBlockRows>>>(src,temp,radius,w,h);

	CUT_CHECK_ERROR("ConvRowGPU() execution failed\n");
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	dim3 blockGridColumns(iDivUp(w, COLUMN_TILE_W),iDivUp(h, COLUMN_TILE_H));
	dim3 threadBlockColumns(COLUMN_TILE_W, 8);

	ConvColGPU<<<blockGridColumns, threadBlockColumns>>>(temp,dest,radius,w,h,COLUMN_TILE_W*8,w*8);

	CUT_CHECK_ERROR("ConvColGPU() execution failed\n");
	CUDA_SAFE_CALL(hipDeviceSynchronize());
}


#define TEST_TILE_W 16
#define TEST_TILE_H 16
#define TEST_THREADS_W 16
#define TEST_THREADS_H 16

__global__ void TestGPU(const float *src, float *dest, int radius, int iters, int width, int height, int step)
{
	__shared__ float pp[4000];
	const int x = __mul24(blockIdx.x,TEST_TILE_W) + threadIdx.x;
	const int y = __mul24(blockIdx.y,TEST_TILE_H) + threadIdx.y;
	int p = __mul24(y, step/sizeof(float)) + x;
	if (x<width-1 && y<height-1 && x>=1 && y>=1)
		{
		float s = src[p],d=s,d2=src[p+1],d3=src[p-1];
		for(int i=0;i<iters;i++)
			if(i%10)
				d = d+d2+d3;
			else 
				for(int j=0;j<10;j++)
					d = (d+d2+d3)/20;
		    //t[pos] += 0.001;
		dest[p] = d;
		}
	__syncthreads();
}

void Test_kernel(const float *src, float *dest, int radius, int iters, int w, int h, int step)
{
	dim3 blockGrid(iDivUp(w,TEST_TILE_W), iDivUp(h,TEST_TILE_H));
	dim3 threadBlock(TEST_THREADS_W,TEST_THREADS_H);

	TestGPU<<<blockGrid, threadBlock>>>(src,dest,radius,iters,w,h,step);

	CUT_CHECK_ERROR("TestGPU() execution failed\n");
	CUDA_SAFE_CALL(hipDeviceSynchronize());
}

